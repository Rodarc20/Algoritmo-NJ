#include "hip/hip_runtime.h"
#include"nj.h"

//este usa una matriz secundaria, don de lamacena la nueva matriz de distancias,ppara luego reemplzar la anteior, habran varios free y malloc
//si es muy lento con tanto malloc, puedo ya que tengo dos arreglos de matiz larga,s cambair entre ellas en cada iteracion y copiar las informacion es de acuerdo al tamño asi evito mover datos, solo copiar a l anueva matrz

/*__device__ inline void swap(float & a, float & b) {
	// Alternative swap doesn't use a temporary register:
	// a ^= b;
	// b ^= a;
	// a ^= b;
    float tmp = a;
    a = b;
    b = tmp;
}

__global__ static void bitonicSort(float * values, int tam) {//modificar para que reciba un a estructura para los minimos
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    // Copy input to shared mem.
    shared[tid] = values[tid];
    __syncthreads();
    // Parallel bitonic sort.
    for (int k = 2; k <= tam; k *= 2) {
        // Bitonic merge:
        for (int j = k / 2; j>0; j /= 2) {
            int ixj = tid ^ j;
            if (ixj > tid) {
                if ((tid & k) == 0) {
                    if (shared[tid] > shared[ixj]) {
                        swap(shared[tid], shared[ixj]);
                    }
                }
                else {
                    if (shared[tid] < shared[ixj]) {
                        swap(shared[tid], shared[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }
    // Write result.
    values[tid] = shared[tid];
}*/

__global__ static void bitonicSortMij(DatosMij * shared, int tam, int DimensionMatrizI) {//modificar para que reciba un a estructura para los minimos
    const int tid = threadIdx.x;
    // Copy input to shared mem.
    __syncthreads();
    // Parallel bitonic sort.
    for (int k = 2; k <= tam; k *= 2) {
        // Bitonic merge:
        for (int j = k / 2; j>0; j /= 2) {
            int ixj = tid ^ j;
            if (ixj > tid) {
                if ((tid & k) == 0) {
                    //if (shared[tid] > shared[ixj]) {
                    if(shared[ixj].Mij < shared[tid].Mij || (shared[ixj].Mij == shared[tid].Mij && shared[tid].prioridad < shared[ixj].prioridad) || (shared[ixj].Mij == shared[tid].Mij && shared[tid].prioridad == shared[ixj].prioridad && shared[ixj].i * DimensionMatrizI + shared[ixj].j < shared[tid].i*DimensionMatrizI + shared[tid].j)){
                    //if(shared[tid].Mij < MMin || (minimos[i].Mij == MMin && prioridadMin < minimos[i].prioridad) || (minimos[i].Mij == MMin && prioridadMin == minimos[i].prioridad && minimos[i].i * DimensionMatrizI + minimos[i].j < iMin*DimensionMatrizI + jMin)){
                        //swap(shared[tid], shared[ixj]);
                        DatosMij temp = shared[tid];
                        shared[tid] = shared[ixj];
                        shared[ixj] = temp;
                    }
                }
                else {
                    //if (shared[tid] < shared[ixj]) {
                    if(shared[tid].Mij < shared[ixj].Mij || (shared[tid].Mij == shared[ixj].Mij && shared[ixj].prioridad < shared[tid].prioridad) || (shared[tid].Mij == shared[ixj].Mij && shared[ixj].prioridad == shared[tid].prioridad && shared[tid].i * DimensionMatrizI + shared[tid].j < shared[ixj].i*DimensionMatrizI + shared[ixj].j)){
                        DatosMij temp = shared[tid];
                        shared[tid] = shared[ixj];
                        shared[ixj] = temp;
                        //swap(shared[tid], shared[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }
}
//dejara el minimo en la primera posicion, en teoria


__global__
void ActualizarArregloId(int * ArregloId, int * ArregloIdNuevo, int DimensionMatrizIN, int i, int j, int NuevoIdJ){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizIN){//o DImensionMatrizI*DimensionMatrizI
        if(k == j){
            ArregloIdNuevo[k] = NuevoIdJ;
        }
        else{
            int aumentok = k >= i;// si tengo valor i, debo copiar el d i+1
            ArregloIdNuevo[k] = ArregloId[k + aumentok];//DimensionMatrizIN+1
        }
    }
}
__global__
void CalcularDivergenciaDevice(float * MatrizDistancia, float * Divergencias, int DimensionActual){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < DimensionActual*DimensionActual){//n es n*n en realidad, estoy recibiendo el valor ya al cuadrado
        //printf("%f\n", Divergencias[i]);
        i /= DimensionActual;
        Divergencias[i] = 0;
        for(int j = 0; j < DimensionActual; j++){
            if(i != j){
                Divergencias[i] += MatrizDistancia[i*DimensionActual+ j];//se toma los nodos reales, para tomar enceuenta los espacios no usados
            }
        }
    }
}
__global__
void CopiarMijDevice(DatosMij * Ordenar, float * MatrizModificada, int *ArregloId, int NumeroNodosReales,  int DimensionMatrizI, int inicialMD, int MC, int TamValores){//funcion ejecutada en la gpu, osea la funcion device
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < TamValores){
        int iM = i / MC;//revisar que estossean los correctos
        int jM = i % MC;
        int imd;
        if(jM/(inicialMD + iM)){
            imd = MC - (inicialMD + iM);
        }
        else{
            imd = inicialMD + iM;
        }
        int jmd = jM % (inicialMD + iM);
        Ordenar[i].Mij = MatrizModificada[i];
        Ordenar[i].i = imd;
        Ordenar[i].j = jmd;
        Ordenar[i].prioridad = (int) (ArregloId[imd]<NumeroNodosReales) + (int) (ArregloId[jmd]< NumeroNodosReales);
    }
}
__global__
void CalcularMijDevice(float * MatrizDistancias, float * Divergencias, float * MatrizModificada, int DimensionMatrizI, int inicialMD, int MC, int TamValores){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < TamValores){
        int iM = i / MC;//revisar que estossean los correctos
        int jM = i % MC;
        int imd;
        if(jM/(inicialMD + iM)){
            imd = MC - (inicialMD + iM);
        }
        else{
            imd = inicialMD + iM;
        }
        int jmd = jM % (inicialMD + iM);
        //int prioridad = (int) Nodos[ArregloId[imd]]->Valido + (int) Nodos[ArregloId[jmd]]->Valido;
        //MatrizDistanciasModificadas[i][j] = Mij(i,j);//no es necasrio almacenar
        //if(MatrizDistanciasModificadas[i][j] <= MMin && prioridadMin < prioridad){//en lugar de matriz deberia se solo una varible temporal
        //printf("%d -- %d: %.5f - %.5f + %.5f\n", imd, jmd, MatrizDistancias[imd*DimensionMatrizI + jmd],Divergencias[imd], Divergencias[jmd]);
        MatrizModificada[i] = MatrizDistancias[imd*DimensionMatrizI + jmd] - (Divergencias[imd] + Divergencias[jmd])/(DimensionMatrizI-2);
        //MatrizModificada[i] = MatrizDistancias[imd*NumeroNodosReales + jmd] - (Divergencias[imd] + Divergencias[jmd])/(DimensionMatrizI-2);
    }
}

__global__
void CalcularNuevaMatrizDevice(float * MatrizDistancias, float * MatrizDistanciasNueva, int DimensionMatrizIN, int i, int j){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizIN * DimensionMatrizIN){//o DImensionMatrizI*DimensionMatrizI
        int iM = k / DimensionMatrizIN;
        int jM = k % DimensionMatrizIN;
        int aumentoI = iM>=i;// si tengo valor i, debo copiar el d i+1
        int aumentoJ = jM>=i;
        if (iM == jM){//diagonal de la nueva matriz
            MatrizDistanciasNueva[iM * DimensionMatrizIN + jM] = 0;
        }
        else if(iM == j || jM == j){//caluclando distancias para el nuevo nodos virtual
            if(iM == j){
                //MatrizDistanciasNueva[iM][jM] = (MatrizDistancias[iM][jM + aumentoJ] + MatrizDistancias[jM + aumentoJ][iM] - Distanciaij)/2;
                //MatrizDistanciasNueva[iM][jM] = (MatrizDistancias[i][jM + aumentoJ] + MatrizDistancias[j][jM + aumentoJ] - MatrizDistancias[i*(DimensionMatrizIN+1) + j])/2;//si la convierto a  matriz compacta debo tener cuiadao
                MatrizDistanciasNueva[iM * DimensionMatrizIN + jM] = (MatrizDistancias[i * (DimensionMatrizIN+1) + jM + aumentoJ] + MatrizDistancias[j*(DimensionMatrizIN+1) + jM + aumentoJ] - MatrizDistancias[i*(DimensionMatrizIN+1) + j])/2;//si la convierto a  matriz compacta debo tener cuiadao
            }
            else{//jM == j
                //MatrizDistanciasNueva[iM][jM] = (MatrizDistancias[iM + aumentoI][jM] + MatrizDistancias[jM][iM + aumentoI] - Distanciaij)/2;
                //MatrizDistanciasNueva[iM][jM] = (MatrizDistancias[iM + aumentoI][i] + MatrizDistancias[iM + aumentoI][j] - MatrizDistancias[i*(DimensionMatrizIN+1) + j])/2;//si la convierto a  matriz compacta debo tener cuiadao
                MatrizDistanciasNueva[iM * DimensionMatrizIN + jM] = (MatrizDistancias[(iM + aumentoI) * (DimensionMatrizIN + 1) + i] + MatrizDistancias[(iM + aumentoI) *(DimensionMatrizIN + 1) + j] - MatrizDistancias[i*(DimensionMatrizIN+1) + j])/2;//si la convierto a  matriz compacta debo tener cuiadao
            }
        }
        else{//traslado a la nueva matriz
            MatrizDistanciasNueva[iM * DimensionMatrizIN + jM] = MatrizDistancias[(iM + aumentoI) * (DimensionMatrizIN + 1) + (jM + aumentoJ)];//DimensionMatrizIN+1
            //MatrizDistanciasNueva[jM * DimensionMatrizIN + iM] = MatrizDistancias[(jM + aumentoJ) * (DimensionMatrizIN + 1) + (iM + aumentoI)];
        }
    }
}

//no estoy usando las siguientes
__global__
void MoviendoIDevice(float * MatrizDistancias, int NumeroNodosReales, int DimensionMatrizI, int i){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizI){
        for(int h = i+1; h < DimensionMatrizI; h++){
            float tempD = MatrizDistancias[(h-1) * NumeroNodosReales + k];
            MatrizDistancias[(h-1)*NumeroNodosReales + k] = MatrizDistancias[h*NumeroNodosReales + k];
            MatrizDistancias[h*NumeroNodosReales + k] = tempD;
        }
    }
}
__global__
void MoviendoJDevice(float * MatrizDistancias, int NumeroNodosReales, int DimensionMatrizI, int i){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizI){
        for(int h = i+1; h < DimensionMatrizI; h++){
            float tempD = MatrizDistancias[k*NumeroNodosReales + h-1];
            MatrizDistancias[k*NumeroNodosReales + h-1] = MatrizDistancias[k*NumeroNodosReales + h];
            MatrizDistancias[k*NumeroNodosReales + h] = tempD;
        }
    }
}
__global__
void ActualizarDistanciasDevice(float * MatrizDistancias, int NumeroNodosReales, int DimensionMatrizI, int j, float Distanciaij){//funcion ejecutada en la gpu, osea la funcion device
    //cout << ini << " - " << fin << endl;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizI){
        //en este punto ya debe estar el nodo virtual nuevo en la posicion i, ya que no se movera alli, quiza yua este antes de entrar a esta funcion
        if(k != j){
            MatrizDistancias[j*NumeroNodosReales + k] = (MatrizDistancias[j*NumeroNodosReales + k] + MatrizDistancias[DimensionMatrizI * NumeroNodosReales + k] - Distanciaij)/2;
            MatrizDistancias[k*NumeroNodosReales + j] = MatrizDistancias[j*NumeroNodosReales + k];//quiza no sea necesario
        }
        else{
            MatrizDistancias[j*NumeroNodosReales + k] = 0;
            MatrizDistancias[k*NumeroNodosReales + j] = 0;
        }
    }
}
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
__global__
void ImprimirFilaDevice(float * FilaDistancia, int n){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        printf("%f\n", FilaDistancia[i]);
    }
}
__global__
void ImprimirDevice(float * MatrizDistancia, int n){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int i = k / n;
    int j = k % n;
    if(k < n*n){
        printf("(%d, %d): %f\n", i, j, MatrizDistancia[i*n + j]);
    }
}

int NJ::GenerarArbol(float ** MatrizDistancia, int NumeroElementos, Nodo ** & Arbol){
    //el primer cambio sera usar un vector que represente la amtriz cuadrada, luego usare un vector para representar la matriz compacat
    //copiar la matriz de distancias//quiza no neceiso modificarla
    //esta amtriz recibida peuede ser la matriz inicial, quiza no sea necesario copiar o almacenarla en esta clase
    DimensionMatrizI= NumeroNodosReales;
    ArregloId = new int [NumeroNodosReales];
    ArregloIdNuevo = new int [NumeroNodosReales];
    Divergencias = new float [NumeroNodosReales];
    MatrizDistancias = new float [NumeroNodosReales * NumeroNodosReales];
    Orden = 0;

    //la primera matriz de distanciasI debe ser igual
    for(int i = 0; i < NumeroNodosReales; i++){//quiza sea util si la matriz no esta de la forma que deseo
        ArregloId[i] = i;
        for(int j = 0; j < NumeroNodosReales; j++){
            MatrizDistancias[i*NumeroNodosReales + j] = MatrizDistancia[i][j];
        }
    }
    //Generando datos en Device
    int sizeMD = NumeroNodosReales*NumeroNodosReales * sizeof(float);
    float *  MatrizDistanciasDevice;
    float *  MatrizDistanciasNuevaDevice;
    hipMalloc((void**) & MatrizDistanciasDevice, sizeMD);
    hipMalloc((void**) & MatrizDistanciasNuevaDevice, sizeMD);
    //desventaja, es que hay que copiarlo todo, incluso los valores uqe ya no se usan
    //lo que podria hacerse es generar una nueva matriz en memoria,y actualzar los datos alli, para luego eliminar la nterior, de esa forma el traspaso se ira haciendo cada vez mas rapido
    hipMemcpy(MatrizDistanciasDevice, MatrizDistancias, sizeMD, hipMemcpyHostToDevice);
    //ImprimirDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasDevice, DimensionMatrizI);

    int sizeDiv = NumeroNodosReales * sizeof(float);
    float * DivergenciasDevice;
    hipMalloc((void **) & DivergenciasDevice , sizeDiv);
    //hipMemcpy(DivergenciasDevice, Divergencias, size, hipMemcpyHostToDevice);

    int inicialMD = (DimensionMatrizI-1)/2 + 1;
    int MC = DimensionMatrizI-1 + !((DimensionMatrizI-1) & 1);
    int Mf = DimensionMatrizI - inicialMD;
    int TamValores = (DimensionMatrizI*DimensionMatrizI - DimensionMatrizI)/2;//(n^2 - n)/2 = 15 ejemplo
    
    int sizeMM = TamValores * sizeof(float);
    MatrizModificada = new float [TamValores];
    float * MatrizModificadaDevice;
    hipMalloc((void **) & MatrizModificadaDevice , sizeMM);

    Ordenar = new DatosMij [TamValores];
    DatosMij * OrdenarDevice;
    int sizeOM = TamValores * sizeof(DatosMij);
    hipMalloc((void **) & OrdenarDevice , sizeOM);

    int * ArregloIdDevice;
    int * ArregloIdNuevoDevice;
    int sizeAID = NumeroNodosReales * sizeof(int);
    hipMalloc((void**) & ArregloIdDevice, sizeAID);
    hipMalloc((void**) & ArregloIdNuevoDevice, sizeAID);
    hipMemcpy(ArregloIdDevice, ArregloId, sizeAID, hipMemcpyHostToDevice);
    //ImprimirArreglo(ArregloId, NumeroNodosReales);
    //ImprimirMatriz(MatrizDistancias, NumeroNodosReales);
    //ImprimirArreglo(ArregloId, NumeroNodosReales);
    //ImprimirMatriz(MatrizDistanciasModificadas, NumeroNodosReales);
    //cout << "copiado de datos correcto" << endl;
    //zona de blce, calcular la matriz de dstancias y luego calcular la matriz de sumas, escoger el menro y calcular la nueva matriz de distancias
    //creacion de matrices
    for(int it = 0; it < NumeroNodosReales-2; it++){//con -2 llego los hago el algoritmo hasta el final
        //bulces
        //cout << "iteracion: " << it << endl;
        inicialMD = (DimensionMatrizI-1)/2 + 1;
        MC = DimensionMatrizI-1 + !((DimensionMatrizI-1) & 1);
        Mf = DimensionMatrizI - inicialMD;
        TamValores = (DimensionMatrizI*DimensionMatrizI - DimensionMatrizI)/2;//(n^2 - n)/2 = 15 ejemplo
        //cout << "inicialMD: " << inicialMD << endl;
        //cout << "MC: " << MC << endl;
        //cout << "Mf: " << Mf << endl;
        //cout << "TamValores: " << TamValores << endl;

        int sizeDivergencias = DimensionMatrizI * sizeof(float);
        CalcularDivergenciaDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasDevice, DivergenciasDevice, DimensionMatrizI);
        hipMemcpy(Divergencias, DivergenciasDevice, sizeDivergencias, hipMemcpyDeviceToHost);//se necesita para la creacion del nuevo nodo virtual
        CalcularMijDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, DivergenciasDevice, MatrizModificadaDevice, DimensionMatrizI, inicialMD, MC, TamValores);
        hipMemcpy(MatrizModificada, MatrizModificadaDevice, TamValores * sizeof(float), hipMemcpyDeviceToHost);
        CopiarMijDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(OrdenarDevice, MatrizModificadaDevice, ArregloIdDevice, NumeroNodosReales, DimensionMatrizI, inicialMD, MC, TamValores);
        bitonicSortMij<<<ceil(TamValores/256.0), 256>>>(OrdenarDevice, TamValores, DimensionMatrizI);
        //bitonicSortMij<<<1, TamValores>>>(OrdenarDevice, TamValores, DimensionMatrizI);
        //bitonicSortMij<<<1, TamValores, TamValores * sizeof(DatosMij)>>>(OrdenarDevice, TamValores, DimensionMatrizI);
        hipMemcpy(Ordenar, OrdenarDevice, TamValores * sizeof(DatosMij), hipMemcpyDeviceToHost);

        //copiar al arreglo de Datos Mij, cuyos i y j, son los convertidos.


        //en realidad esta busuqeda del minimo deberia estar en cuda
        /*int iMin = DimensionMatrizI, jMin = DimensionMatrizI;
        int imdMin = DimensionMatrizI, jmdMin = DimensionMatrizI;
        int prioridadMin = -1;
        float MMin = numeric_limits<float>::max();//valor maximo
        //copiada quiza no sea util
        //cout << "Minimos" << endl;
        for(int i = 0; i < Mf; i++){//la forma de recorrer hace que i > j siempre primera posicion (1,0)
            for(int j = 0; j < MC; j++){//para recorrer la matriz tringula inferior
                //if(i != j){//ya no es necesario este if
                //arreglar lo de la prioirdad no son u y j, son ostros valores
                //comparar con los md o con los i j dde la matriz modificada?
                    int imd;
                    if(j/(inicialMD + i)){
                        imd = MC - (inicialMD + i);
                    }
                    else{
                        imd = inicialMD + i;
                    }
                    int jmd = j % (inicialMD + i);
                    int prioridad = (int) Nodos[ArregloId[imd]]->Valido + (int) Nodos[ArregloId[jmd]]->Valido;
                    //MatrizDistanciasModificadas[i][j] = Mij(i,j);//no es necasrio almacenar
                    //if(MatrizDistanciasModificadas[i][j] <= MMin && prioridadMin < prioridad){//en lugar de matriz deberia se solo una varible temporal
                    //if(ActualMij <= MMin && prioridadMin < prioridad){//en lugar de matriz deberia se solo una varible temporal
                    //cout << i << " " << j << ": " << MatrizModificada[i*MC + j] << " (" << prioridad << ")" << "[" << imd * DimensionMatrizI + j << "]" << endl;
                    //if(MatrizModificada[i*MC + j] < MMin || (MatrizModificada[i*MC + j] == MMin && prioridadMin < prioridad) || (MatrizModificada[i*MC + j] == MMin && prioridadMin == prioridad && imd*MC + jmd < iMin*MC + jMin)){
                    //if(MatrizModificada[i*MC + j] < MMin || (MatrizModificada[i*MC + j] == MMin && prioridadMin < prioridad)){
                    if(MatrizModificada[i*MC + j] < MMin || (MatrizModificada[i*MC + j] == MMin && prioridadMin < prioridad) || (MatrizModificada[i*MC + j] == MMin && prioridadMin == prioridad && imd * DimensionMatrizI + jmd < imdMin*DimensionMatrizI + jmdMin)){
                    //if(MatrizDistanciasModificadas[i][j] < MMin){//en lugar de matriz deberia se solo una varible temporal
                        iMin = i;
                        jMin = j;
                        imdMin = imd;
                        jmdMin = jmd;
                        MMin = MatrizModificada[i*MC + j];
                        //MMin = MatrizDistanciasModificadas[i][j];
                        prioridadMin = prioridad;
                    }
                //}
            }
        }*/
        //cout << "eleccion del modificado sij " << iMin << " " << jMin << endl;//no esta mostrando el nombre, esta mostrando los i j de la matriz
        /*int imd;
        if(jMin/(inicialMD + iMin)){
            imd = MC - (inicialMD + iMin);
        }
        else{
            imd = inicialMD + iMin;
        }
        int jmd = jMin % (inicialMD + iMin);
        iMin = imd;
        jMin = jmd;*/

        /*for(int i = 0; i < TamValores; i++){
            cout << i << ": " << Ordenar[i].Mij << " " << Ordenar[i].i << " " << Ordenar[i].j << " " << Ordenar[i].prioridad << endl;
        }*/
        int iMin = Ordenar[0].i;
        int jMin = Ordenar[0].j;
        if(it == NumeroNodosReales-3){//este caso deberia estar mejo estructurado
            //cout << "caso especial: " << it << endl;
            float DMin = numeric_limits<float>::max();//valor maximo
            //copiada quiza no sea util
            for(int i = 1; i < DimensionMatrizI; i++){
                for(int j = 0; j < i; j++){
                    //no calcular si i y j son iguales; y no hacer caluclo doble, es decir solo recorrel el triangulo superior o inferior
                    //if(i != j){
                        if(MatrizDistancias[i*NumeroNodosReales + j] < DMin){//en lugar de matriz deberia se solo una varible temporal
                            iMin = i;
                            jMin = j;
                            DMin = MatrizDistancias[i*NumeroNodosReales + j];
                        }
                    //}
                }
            }
        }
        //una vez seleccionados el i y j , deberia solo ahcer un cambio para que esto funcione sin mover las otras funciones, o hacer el cambio definitivo, ya que esteo siempre se cumplira!!, lo de que j sea mayo que i


        //cout << "distancias" << endl;
        //ImprimirArreglo(ArregloId, DimensionMatrizI);
        //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
        //ImprimirDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasDevice, DimensionMatrizI);
        //cout << "distancias modificadas" << endl;
        //ImprimirArreglo(ArregloId, DimensionMatrizI);
        //ImprimirMatriz(MatrizDistanciasModificadas, DimensionMatrizI);
        //cout << "eleccion del sij " << ArregloId[iMin] << " " << ArregloId[jMin] << endl;//no esta mostrando el nombre, esta mostrando los i j de la matriz
        //ya tengo los nodos mas similares
        //crear un nuevo nodo virtual, reemplazar
        //cout << "eleccion del sij " << iMin << "(" << ArregloId[iMin] << ") - " << jMin << "(" << ArregloId[jMin] << ")" << endl;//no esta mostrando el nombre, esta mostrando los i j de la matriz
        CrearNodoVirtual(iMin, jMin);
        //CrearNodoVirtual(ArregloId[iMin], ArregloId[jMin]);
        //cout << "creado nodo virtual" << endl;
        //substituimos el ide del nodo virrual en el i
        
        //ArregloId[iMin] = NumeroNodos-1;//como  j tiene que ser mayo que i e cambiado
        //ArregloId[jMin] = NumeroNodos-1;//como  j tiene que ser mayo que i e cambiado
        //esto deberia ser el numero nodo actual
        //jArregloId[jMin] = NumeroNodosReales + NumeroNodosVirtuales - 1;//estara dentro de la funcion kernel
        //ImprimirArreglo(ArregloId, DimensionMatrizI);

        DimensionMatrizI--;

        //ArregloId[jMin] = NumeroNodosReales + NumeroNodosVirtuales - 1;//estara dentro de la funcion kernel
        ActualizarArregloId<<<ceil(DimensionMatrizI/256.0), 256>>>(ArregloIdDevice, ArregloIdNuevoDevice, DimensionMatrizIN, iMin, jMin, NumeroNodosReales + NumeroNodosVirtuales - 1);
        int * AIDTemp = ArregloIdDevice;
        ArregloIdDevice = ArregloIdNuevoDevice;
        ArregloIdNuevoDevice = AIDTemp;
        hipMemcpy(ArregloId, ArregloIdDevice, sizeAID, hipMemcpyDeviceToHost);//actualzar ese size

        CalcularNuevaMatrizDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, MatrizDistanciasNuevaDevice, DimensionMatrizI, iMin, jMin);
        float * MDTemp = MatrizDistanciasDevice;
        MatrizDistanciasDevice = MatrizDistanciasNuevaDevice;
        MatrizDistanciasNuevaDevice = MDTemp;
        sizeMD = DimensionMatrizI * DimensionMatrizI * sizeof(float);
        hipMemcpy(MatrizDistancias, MatrizDistanciasDevice, sizeMD, hipMemcpyDeviceToHost);
        //cout << "Nueva distancias" << endl;
        //ImprimirArreglo(ArregloId, DimensionMatrizI);
        //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
        //NuevaMatrizDistancias(iMin, jMin, MatrizDistanciasDevice);//ya se redujo el tamaño de la matriz//aqui dentro esta el n--
        //cout << "creada nueva matriz de distancias" << endl;
    }
    //al final se calculo una nuev matriz de distancias, por lo tanto arreglo ID tiene solo 2 elementos
    //UNION: a ellos los unimos
    //cout << "final de las iteracion hasta n-2" << endl;
    //cout << "distancias" << endl;
    //ImprimirArreglo(ArregloId, DimensionMatrizI);
    //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
    //cout << "distancias modificadas" << endl;
    //ImprimirArreglo(ArregloId, DimensionMatrizI);
    //ImprimirMatriz(MatrizDistanciasModificadas, DimensionMatrizI);
    Nodos[ArregloId[0]]->Padre = Nodos[ArregloId[1]];
    Nodos[ArregloId[0]]->PadreId = ArregloId[1];//o usando -D sobre la linea anterior
    Nodos[ArregloId[0]]->DistanciaPadre= MatrizDistancias[0*NumeroNodosReales + 1];//o usando -D sobre la linea anterior
    Nodos[ArregloId[1]]->Padre = Nodos[ArregloId[0]];
    Nodos[ArregloId[1]]->PadreId = ArregloId[0];//o usando -D sobre la linea anterior
    Nodos[ArregloId[1]]->DistanciaPadre= MatrizDistancias[0*NumeroNodosReales + 1];//o usando -D sobre la linea anterior
    //falta calcular estas distancias
    Arbol = Nodos;
    hipFree(MatrizDistanciasDevice);
    hipFree(MatrizDistanciasNuevaDevice);
    hipFree(DivergenciasDevice);
    hipFree(ArregloIdDevice);
    hipFree(ArregloIdNuevoDevice);
    return NumeroNodos;
}

float NJ::Mij(int i, int j){// en espacio de arreglo id 
    //como trabajo sobre la matriz inferior, esta bien lo i j que recibo, que me marque el (1,0)
    return MatrizDistancias[i*NumeroNodosReales + j] - (Divergencias[i] + Divergencias[j])/(DimensionMatrizI-2);
}

void NJ::NuevaMatrizDistancias(int i, int j, float * MatrizDistanciasDevice){//i y j son los nodos que fueron desginados como similares en esa iteracion
    //antes de ahcer el corrimiento seria buno calcular las deistancias para el nuevo, y solo hacer el cambio con i
    //como el j es el mas a la izquierda, ese lo dejo, y muevo el i hacia el final de la matriz
    //Host solo almacena los punteros de las filas en device
    //cout << "Nueva matriz de distacnias " << i << ", " << j << endl;//se escojo e primero
    float Distanciaij = MatrizDistancias[i*NumeroNodosReales + j];//este se puede dejar asi, ya que uso la matriz inferior

    for(int k = i+1; k < DimensionMatrizI; k++){
        int tempId = ArregloId[k-1];//solo debe modificar lo concerniete al arreglo j
        ArregloId[k-1] = ArregloId[k];//a la vez que cambio debo actulizar sus poseisiones en el arregloID
        ArregloId[k] = tempId;
        
    }
    //estos i y j, estan sobre el ArrgloId
    MoviendoIDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, NumeroNodosReales, DimensionMatrizI, i);
    MoviendoJDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, NumeroNodosReales, DimensionMatrizI, i);
    //cout << "i al final" << endl;
    //ImprimirArreglo(ArregloId, DimensionMatrizI);
    //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
    DimensionMatrizI--;
    //j ahora esta en la posicion DimensionMatriz
    ActualizarDistanciasDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, NumeroNodosReales, DimensionMatrizI, j, Distanciaij);
    int size = NumeroNodosReales * NumeroNodosReales * sizeof(float);
    //int size = DimensionMatrizI * sizeof(float);
    hipMemcpy(MatrizDistancias, MatrizDistanciasDevice, size, hipMemcpyDeviceToHost);
    //cout << "fin creacio nnueva matriz" << endl;
    //en que momento ubico al nuevo nodo viertual cundo lo creo?
    //
}

void NJ::ImprimirMatriz(float * Matriz, int n){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            printf("%.3f\t", Matriz[i * n + j]);
        }
        printf("\n");
    }
}

void NJ::ImprimirArreglo(int * Arreglo, int n){
    for(int i = 0; i < n; i++){
        printf("%d\t", ArregloId[i]);
    }
    printf("\n");
}

void NJ::ImprimirDivergencias(float * Divergencia, int n){
    printf("Divergencias:\n");
    for(int i = 0; i < n; i++){
        printf("%.3f\t", Divergencia[i]);
    }
    printf("\n");
}

void NJ::CrearNodoVirtual(int i, int j){//es que deberia recibir algun dato, nombre del documento quiza, para los reales
    //que pasa si quiero crear mas nodos reales? mm en teoria no puedo, ya que habira que recalcular todo
    //creo que esto sera para crear nodos virtuales
    ////recibe los i y j del arreglo de nodos
    //aqui recibo los i y j siendo i > j, por lo tanto lo que hacia con j debo hacerlo con y y lo que hacia con i debo hacerlo conj
    //float Lj = MatrizDistancias[j][i]/2 + (Divergencias[j] - Divergencias[i])/(2*(DimensionMatrizI-2));
    //float Li = MatrizDistancias[j][i] - Lj;
    //cout << "entrada: " << i << "-" << j << endl;
    //cout << MatrizDistancias[i * DimensionMatrizI + j] << endl;
    //cout << Divergencias[j] << endl;
    //cout << Divergencias[i] << endl;
    float Lj = MatrizDistancias[i*DimensionMatrizI+ j]/2 + (Divergencias[j] - Divergencias[i])/(2*(DimensionMatrizI-2));
    float Li = MatrizDistancias[i*DimensionMatrizI+ j] - Lj;
    i = ArregloId[i];
    j = ArregloId[j];

    int nv = NumeroNodosReales + NumeroNodosVirtuales;
    //cout <<"arreglo: " << i << "-" << j << " " << "id nuevo nodo virtual " << nv << endl;
    //cout << "Lj: " << Lj << " Li: " << Li << endl;

    //Nodos[nv]->Id = nv;
    Nodos[nv]->HijosId[0] = j;
    Nodos[nv]->HijosId[1] = i;
    Nodos[nv]->Hijos[0] = Nodos[j];
    Nodos[nv]->Hijos[1] = Nodos[i];
    Nodos[nv]->DistanciasHijos[0] = Lj;
    Nodos[nv]->DistanciasHijos[1] = Li;
    Nodos[nv]->Orden = 0;//iteracion
    Nodos[j]->Padre = Nodos[nv];
    Nodos[j]->PadreId = Nodos[nv]->Id;//o odria ser solo NumeroNodos
    Nodos[j]->DistanciaPadre = Lj;
    Nodos[j]->Orden = Orden;//iteracion
    Orden++;
    Nodos[i]->Padre = Nodos[nv];
    Nodos[i]->PadreId = Nodos[nv]->Id;//o odria ser solo NumeroNodos
    Nodos[i]->DistanciaPadre = Li;
    Nodos[i]->Orden = Orden;//iteracion
    Orden++;
    NumeroNodosVirtuales++;

    //falta agregar lo de las ditancias de los branchs
//afuera de esta funcon se maneja lo de las hojas para los nodos virtaules, 
}

//mejorar las contruscciones de los nodos por defecto y demas, para hacer mas eficiente esta arte
//y si uso la clase vector?? contruir una version que lo use
void NJ::DatosIniciales(string * Datos, int n){
    //en teoria siempre se necesitan la misma cantidad de nodos virtuales, ya que el algoritmos siempre da las iteraciones fijas, solo depende de si n es par o impar;
    //por lo lanto se pueden crear de una vez los nodos virtuales, y solo llenarlos con la impofrmacion pertinenete y determnando a lcula llear, en cada iteracion, en lugar de estar creandolos en cada iteracion
    //crear los nodos reales
    NumeroNodos = n + n - 2;
    NumeroNodosReales = n;
    Nodos = new Nodo * [NumeroNodos];
    for(int i = 0; i < NumeroNodosReales; i++){
        Nodos[i] = new Nodo;
        Nodos[i]->Id = i;
        Nodos[i]->Valido = 1;//si sera virtual o real
        Nodos[i]->Nombre = Datos[i];
    }
    for(int i = NumeroNodosReales; i < NumeroNodos; i++){//para crear los nodos virtuales
        Nodos[i] = new Nodo;
        Nodos[i]->Id = i;
        Nodos[i]->Valido = 0;//si sera virtual o real
    }
    NumeroNodosVirtuales = 0;//aun no los he creado coomo parte del algoritmo,
    //esta variable me ayudara a iterar para trabajr sobre le nodo virtual aporpiado,
    //lo que nates hacian cuando incrementaba la variable NUmeroNodos en cada iteracion
}

NJ::NJ(){

}

NJ::~NJ(){
    delete [] MatrizDistancias;
    delete [] ArregloId;
    delete [] Divergencias;
    //el arreglo nodos debo eliminarlo??
}
//Probar vreando una nueva matriz para la nueva matriz de distancias, este calculo es en paralelo, y ya no habria corrimientos, bastaria con calcular las nuevas posiciones dentro de cada thread
//asi copiar menos informacion

//buscar donde hay redundancia en el acceso a datos, quiza todos los sij, se puedan calcular a la vez, o cosas asi
//busar como manejar en la misma matriz los nuevos nodos virtuales, los reemplazos etc, para no crear matrices mas grandes, y seguir utilizando la que esta, quiza ids temporanoles, o que se yo
////la cantidad de iteraciones es fija, por lo tanto crear los nodos virtuales necesarios desde el inicio, y en la funcion CrearNodoVIrual solamente hacer las uniones y calculos respectivos
