#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<string>
#include<ctime>
#include"nj.h"
#include"nodo.h"

using namespace std;

void ImprimirNodos(Nodo ** nodos, int n){
    for(int i = 0; i < n; i++){
        cout << "Id: " << nodos[i]->Id << endl;
        cout << "Nombre: " << nodos[i]->Nombre << endl;
        cout << "Padre: " << nodos[i]->PadreId << " -- " << nodos[i]->DistanciaPadre << endl;
        cout << "Hijo0: " << nodos[i]->HijosId[0] << " -- " << nodos[i]->DistanciasHijos[0] << endl;
        cout << "Hijo1: " << nodos[i]->HijosId[1] << " -- " << nodos[i]->DistanciasHijos[1] << endl;
    }
}

void ImprimirNodosPex(Nodo ** nodos, int n){
    cout << "<?xml version=\"1.0\" encoding=\"ISO-8859-1\" ?>" << endl;
    cout << "<graph description=\"\">" << endl;

    for(int i = 0; i < n; i++){
        cout << "<vertex id=\"" << nodos[i]->Id << "\">" << endl;
        cout << "<valid value=\"" << nodos[i]->Valido << "\"/>" << endl;
        cout << "<x-coordinate value=\"" << i*6 << "\"/>" << endl;
        cout << "<y-coordinate value=\"" << i*6 << "\"/>" << endl;
        cout << "<url value=\"" << "\"/>" << endl;
        cout << "<order value=\"" << nodos[i]->Orden << "\"/>" << endl;//no esta en el pex
        cout << "<scalars>" << endl;
        cout << "<scalar name=\"cdata\" value=\"" << 0.0 << "\"/>" << endl;
        cout << "</scalars>" << endl;
        cout << "<labels>" << endl;
        cout << "<label name=\"title\" value=\"" << nodos[i]->Nombre << "\"/>" << endl;
        cout << "<label name=\"file name\" value=\"" << "\"/>" << endl;
        cout << "</labels>" << endl;
        if(nodos[i]->HijosId[0] != -1){
            cout << "<son value=\"" << nodos[i]->HijosId[0] << "\" distance=\"" << nodos[i]->DistanciasHijos[0] << "\"/>"<< endl;
        }
        if(nodos[i]->HijosId[1] != -1){
            cout << "<son value=\"" << nodos[i]->HijosId[1] << "\" distance=\"" << nodos[i]->DistanciasHijos[1] << "\"/>"<< endl;
        }
        cout << "<parent value=\"" << nodos[i]->PadreId << "\" distance=\"" << nodos[i]->DistanciaPadre << "\"/>"<< endl;
        cout << "</vertex>" << endl;
    }
    cout << "</graph>" << endl;
}

void LeerDatos(float ** & m, string * & d, int & n){//lee matrices cuadrasdas con  todos su valores, primero el numero de datos, luego los nombres de los datos y luego la matriz
    cin >> n;
    m = new float * [n];
    d = new string [n];
    for(int i = 0; i < n; i++){
        cin >> d[i];
        m[i] = new float [n];
    }
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            cin >> m[i][j];
        }
    }
}

void LeerDatosSinNombre(float ** & m, string * & d, int & n){//lee matrices cuadrasdas con  todos su valores, primero el numero de datos, luego los nombres de los datos y luego la matriz
    cin >> n;
    m = new float * [n];
    d = new string [n];
    for(int i = 0; i < n; i++){
        d[i] = i;
        m[i] = new float [n];
    }
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            cin >> m[i][j];
        }
    }
}
void LeerDatosPex(float ** & m, string * & d, int & n){//lee matrices cuadrasdas con  todos su valores, primero el numero de datos, luego los nombres de los datos, luego la clase de los datos para el cdata y luego la matriz
    
    cin >> n;
    m = new float * [n];
    d = new string [n];
    string * clase = new string [n];
    for(int i = 0; i < n; i++){
        cin >> d[i];
        m[i] = new float [n];
    }
    for(int i = 0; i < n; i++){
        cin >> clase[i];
    }
    for(int i = 1; i < n; i++){
        for(int j = 0; j < i; j++){
            cin >> m[i][j];
            m[j][i] = m[i][j];//esto se queita si solo quiero almacenar la matriz inferior
        }
    }
    //devuelvo matrices cuadradas, con los datos reflejados
    delete [] clase;
}

int main(){
    unsigned t0,t1;
    int n;
    float ** m;
    string * d;
    /*cin >> n;
    float ** m = new float * [n];
    string * d = new string [n];
    for(int i = 0; i < n; i++){
        cin >> d[i];
        m[i] = new float [n];
    }
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            cin >> m[i][j];
        }
    }*/
    //LeerDatosPex(m, d, n);
    LeerDatosSinNombre(m, d, n);
    NJ nj;
    Nodo ** result;
    nj.DatosIniciales(d, n);
    t0 = clock();
    int tam = nj.GenerarArbol(m, n, result);//en toroia no encesitorecibir n ya que ya lo recibo en datos iniciales
    t1 = clock();
    //ImprimirNodos(result, tam);
    ImprimirNodosPex(result, tam);
    double time = (double(t1-t0)/CLOCKS_PER_SEC);
    printf("Finalizado: %f\n", time);
    for(int i = 0; i < n; i++){
        delete [] m[i];
    }
    delete [] m;
    delete [] d;
    for(int i = 0; i < tam; i++){
        delete result[i];
    }
    delete [] result;
    return 0;
}
