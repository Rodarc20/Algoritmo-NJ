#include "hip/hip_runtime.h"
#include"nj.h"

__global__
void CalcularDivergenciaDevice(float ** MatrizDistancia, float * Divergencias, int n){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        //printf("%f\n", Divergencias[i]);
        Divergencias[i] = 0;
        for(int j = 0; j < n; j++){
            if(i != j){
                Divergencias[i] += MatrizDistancia[i][j];
            }
        }
    }
}
__global__
void CalcularMijDevice(float ** MatrizDistancias, float * Divergencias, float * MatrizModificada, int DimensionMatrizI, int inicialMD, int MC, int TamValores){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < TamValores){
        int iM = i / MC;//revisar que estossean los correctos
        int jM = i % MC;
        int imd;
        if(jM/(inicialMD + iM)){
            imd = MC - (inicialMD + iM);
        }
        else{
            imd = inicialMD + iM;
        }
        int jmd = jM % (inicialMD + iM);
        //int prioridad = (int) Nodos[ArregloId[imd]]->Valido + (int) Nodos[ArregloId[jmd]]->Valido;
        //MatrizDistanciasModificadas[i][j] = Mij(i,j);//no es necasrio almacenar
        //if(MatrizDistanciasModificadas[i][j] <= MMin && prioridadMin < prioridad){//en lugar de matriz deberia se solo una varible temporal
        MatrizModificada[i] = MatrizDistancias[imd][jmd] - (Divergencias[imd] + Divergencias[jmd])/(DimensionMatrizI-2);
    }
}
__global__
void MoviendoIDevice(float ** MatrizDistancias, int DimensionMatrizI, int i){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizI){
        for(int h = i+1; h < DimensionMatrizI; h++){
            float tempD = MatrizDistancias[h-1][k];
            MatrizDistancias[h-1][k] = MatrizDistancias[h][k];
            MatrizDistancias[h][k] = tempD;
        }
    }
}
__global__
void MoviendoJDevice(float ** MatrizDistancias, int DimensionMatrizI, int i){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizI){
        for(int h = i+1; h < DimensionMatrizI; h++){
            float tempD = MatrizDistancias[k][h-1];
            MatrizDistancias[k][h-1] = MatrizDistancias[k][h];
            MatrizDistancias[k][h] = tempD;
        }
    }
}
__global__
void ActualizarDistanciasDevice(float ** MatrizDistancias, int DimensionMatrizI, int j, float Distanciaij){//funcion ejecutada en la gpu, osea la funcion device
    //cout << ini << " - " << fin << endl;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < DimensionMatrizI){
        //en este punto ya debe estar el nodo virtual nuevo en la posicion i, ya que no se movera alli, quiza yua este antes de entrar a esta funcion
        if(k != j){
            MatrizDistancias[j][k] = (MatrizDistancias[j][k] + MatrizDistancias[DimensionMatrizI][k] - Distanciaij)/2;
            MatrizDistancias[k][j] = MatrizDistancias[j][k];//quiza no sea necesario
        }
        else{
            MatrizDistancias[j][k] = 0;
            MatrizDistancias[k][j] = 0;
        }
    }
}
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
__global__
void ImprimirFilaDevice(float * FilaDistancia, int n){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        printf("%f\n", FilaDistancia[i]);
    }
}
__global__
void ImprimirDevice(float ** MatrizDistancia, int n){//funcion ejecutada en la gpu, osea la funcion device
    //n es la Dimension de la matriz y naturalmente tambien de las divergencias
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int i = k / n;
    int j = k % n;
    if(k < n*n){
        printf("(%d, %d): %f\n", i, j, MatrizDistancia[i][j]);
    }
}

int NJ::GenerarArbol(float ** MatrizDistancia, int NumeroElementos, Nodo ** & Arbol){
    //copiar la matriz de distancias//quiza no neceiso modificarla
    //esta amtriz recibida peuede ser la matriz inicial, quiza no sea necesario copiar o almacenarla en esta clase
    DimensionMatrizI= NumeroNodosReales;
    ArregloId = new int [NumeroNodosReales];
    Divergencias = new float [NumeroNodosReales];
    MatrizDistancias = new float * [NumeroNodosReales];
    Orden = 0;

    //la primera matriz de distanciasI debe ser igual
    for(int i = 0; i < NumeroNodosReales; i++){//quiza sea util si la matriz no esta de la forma que deseo
        MatrizDistancias[i] = new float [NumeroNodosReales];
        //MatrizDistanciasModificadas[i] = new float [NumeroElementos];
        ArregloId[i] = i;
        //Divergencias[i] = i;
        for(int j = 0; j < NumeroNodosReales; j++){
            MatrizDistancias[i][j] = MatrizDistancia[i][j];
        }
    }
    //Generando datos en Device
    int size = NumeroNodosReales * sizeof(float);
    int sizem = NumeroNodosReales * sizeof(float*);
    float **  MatrizDistanciasHost = new float * [NumeroNodosReales];//para controlar la creacion de memoria en device
    float **  MatrizDistanciasDevice;
    hipMalloc((void***) & MatrizDistanciasDevice, sizem);
    for(int i = 0; i < NumeroNodosReales; i++){
        hipMalloc((void **) &(MatrizDistanciasHost[i]), size);
        hipMemcpy(MatrizDistanciasHost[i], MatrizDistancias[i], size, hipMemcpyHostToDevice);
        //ImprimirFilaDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasHost[i], DimensionMatrizI);
    }
    hipMemcpy(MatrizDistanciasDevice, MatrizDistanciasHost, sizem, hipMemcpyHostToDevice);
    //ImprimirDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasDevice, DimensionMatrizI);

    float * DivergenciasDevice;
    hipMalloc((void **) & DivergenciasDevice , size);
    //hipMemcpy(DivergenciasDevice, Divergencias, size, hipMemcpyHostToDevice);

    int inicialMD = (DimensionMatrizI-1)/2 + 1;
    int MC = DimensionMatrizI-1 + !((DimensionMatrizI-1) & 1);
    int Mf = DimensionMatrizI - inicialMD;
    int TamValores = (DimensionMatrizI*DimensionMatrizI - DimensionMatrizI)/2;//(n^2 - n)/2 = 15 ejemplo
    
    size = TamValores * sizeof(float);
    MatrizModificada = new float [TamValores];
    float * MatrizModificadaDevice;
    hipMalloc((void **) & MatrizModificadaDevice , size);

    //ImprimirArreglo(ArregloId, NumeroNodosReales);
    //ImprimirMatriz(MatrizDistancias, NumeroNodosReales);
    //ImprimirArreglo(ArregloId, NumeroNodosReales);
    //ImprimirMatriz(MatrizDistanciasModificadas, NumeroNodosReales);
    //cout << "copiado de datos correcto" << endl;
    //zona de blce, calcular la matriz de dstancias y luego calcular la matriz de sumas, escoger el menro y calcular la nueva matriz de distancias
    //creacion de matrices
    for(int it = 0; it < NumeroNodosReales-2; it++){//con -2 llego los hago el algoritmo hasta el final
        //bulces
        //cout << "iteracion: " << it << endl;
        inicialMD = (DimensionMatrizI-1)/2 + 1;
        MC = DimensionMatrizI-1 + !((DimensionMatrizI-1) & 1);
        Mf = DimensionMatrizI - inicialMD;
        TamValores = (DimensionMatrizI*DimensionMatrizI - DimensionMatrizI)/2;//(n^2 - n)/2 = 15 ejemplo
        //cout << "inicialMD: " << inicialMD << endl;
        //cout << "MC: " << MC << endl;
        //cout << "Mf: " << Mf << endl;
        //cout << "TamValores: " << TamValores << endl;

        int sizeDivergencias = DimensionMatrizI * sizeof(float);
        CalcularDivergenciaDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasDevice, DivergenciasDevice, DimensionMatrizI);
        hipMemcpy(Divergencias, DivergenciasDevice, sizeDivergencias, hipMemcpyDeviceToHost);//se necesita para la creacion del nuevo nodo virtual
        CalcularMijDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, DivergenciasDevice, MatrizModificadaDevice, DimensionMatrizI, inicialMD, MC, TamValores);
        hipMemcpy(MatrizModificada, MatrizModificadaDevice, TamValores * sizeof(float), hipMemcpyDeviceToHost);


        //en realidad esta busuqeda del minimo deberia estar en cuda
        int iMin = DimensionMatrizI, jMin = DimensionMatrizI;
        int imdMin = DimensionMatrizI, jmdMin = DimensionMatrizI;
        int prioridadMin = -1;
        float MMin = numeric_limits<float>::max();//valor maximo
        //copiada quiza no sea util
        for(int i = 0; i < Mf; i++){//la forma de recorrer hace que i > j siempre primera posicion (1,0)
            for(int j = 0; j < MC; j++){//para recorrer la matriz tringula inferior
                //if(i != j){//ya no es necesario este if
                //arreglar lo de la prioirdad no son u y j, son ostros valores
                //comparar con los md o con los i j dde la matriz modificada?
                    int imd;
                    if(j/(inicialMD + i)){
                        imd = MC - (inicialMD + i);
                    }
                    else{
                        imd = inicialMD + i;
                    }
                    int jmd = j % (inicialMD + i);
                    int prioridad = (int) Nodos[ArregloId[imd]]->Valido + (int) Nodos[ArregloId[jmd]]->Valido;
                    //MatrizDistanciasModificadas[i][j] = Mij(i,j);//no es necasrio almacenar
                    //if(MatrizDistanciasModificadas[i][j] <= MMin && prioridadMin < prioridad){//en lugar de matriz deberia se solo una varible temporal
                    //if(ActualMij <= MMin && prioridadMin < prioridad){//en lugar de matriz deberia se solo una varible temporal
                    //cout << i << " " << j << ": " << MatrizModificada[i*MC + j] << endl;
                    //if(MatrizModificada[i*MC + j] < MMin || (MatrizModificada[i*MC + j] == MMin && prioridadMin < prioridad) || (MatrizModificada[i*MC + j] == MMin && prioridadMin == prioridad && imd*MC + jmd < iMin*MC + jMin)){
                    //if(MatrizModificada[i*MC + j] < MMin || (MatrizModificada[i*MC + j] == MMin && prioridadMin < prioridad)){
                    if(MatrizModificada[i*MC + j] < MMin || (MatrizModificada[i*MC + j] == MMin && prioridadMin < prioridad) || (MatrizModificada[i*MC + j] == MMin && prioridadMin == prioridad && imd * DimensionMatrizI + jmd < imdMin*DimensionMatrizI + jmdMin)){
                    //if(MatrizDistanciasModificadas[i][j] < MMin){//en lugar de matriz deberia se solo una varible temporal
                        iMin = i;
                        jMin = j;
                        imdMin = imd;
                        jmdMin = jmd;
                        MMin = MatrizModificada[i*MC + j];
                        //MMin = MatrizDistanciasModificadas[i][j];
                        prioridadMin = prioridad;
                    }
                //}
            }
        }
        //cout << "eleccion del modificado sij " << iMin << " " << jMin << endl;//no esta mostrando el nombre, esta mostrando los i j de la matriz
        int imd;
        if(jMin/(inicialMD + iMin)){
            imd = MC - (inicialMD + iMin);
        }
        else{
            imd = inicialMD + iMin;
        }
        int jmd = jMin % (inicialMD + iMin);
        iMin = imd;
        jMin = jmd;
        if(it == NumeroNodosReales-3){//este caso deberia estar mejo estructurado
            //cout << "caso especial: " << it << endl;
            float DMin = numeric_limits<float>::max();//valor maximo
            //copiada quiza no sea util
            for(int i = 1; i < DimensionMatrizI; i++){
                for(int j = 0; j < i; j++){
                    //no calcular si i y j son iguales; y no hacer caluclo doble, es decir solo recorrel el triangulo superior o inferior
                    //if(i != j){
                        if(MatrizDistancias[i][j] < DMin){//en lugar de matriz deberia se solo una varible temporal
                            iMin = i;
                            jMin = j;
                            DMin = MatrizDistancias[i][j];
                        }
                    //}
                }
            }
        }
        //una vez seleccionados el i y j , deberia solo ahcer un cambio para que esto funcione sin mover las otras funciones, o hacer el cambio definitivo, ya que esteo siempre se cumplira!!, lo de que j sea mayo que i


        //cout << "distancias" << endl;
        //ImprimirArreglo(ArregloId, DimensionMatrizI);
        //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
        //ImprimirDevice<<<ceil(DimensionMatrizI/256.0), 256>>> (MatrizDistanciasDevice, DimensionMatrizI);
        //cout << "distancias modificadas" << endl;
        //ImprimirArreglo(ArregloId, DimensionMatrizI);
        //ImprimirMatriz(MatrizDistanciasModificadas, DimensionMatrizI);
        //cout << "eleccion del sij " << ArregloId[iMin] << " " << ArregloId[jMin] << endl;//no esta mostrando el nombre, esta mostrando los i j de la matriz
        //ya tengo los nodos mas similares
        //crear un nuevo nodo virtual, reemplazar
        //cout << "eleccion del sij " << iMin << "(" << ArregloId[iMin] << ") - " << jMin << "(" << ArregloId[jMin] << ")" << endl;//no esta mostrando el nombre, esta mostrando los i j de la matriz
        CrearNodoVirtual(iMin, jMin);
        //CrearNodoVirtual(ArregloId[iMin], ArregloId[jMin]);
        //cout << "creado nodo virtual" << endl;
        //substituimos el ide del nodo virrual en el i
        
        //ArregloId[iMin] = NumeroNodos-1;//como  j tiene que ser mayo que i e cambiado
        //ArregloId[jMin] = NumeroNodos-1;//como  j tiene que ser mayo que i e cambiado
        //esto deberia ser el numero nodo actual
        ArregloId[jMin] = NumeroNodosReales + NumeroNodosVirtuales - 1;
        //ImprimirArreglo(ArregloId, DimensionMatrizI);

        NuevaMatrizDistancias(iMin, jMin, MatrizDistanciasDevice, MatrizDistanciasHost);//ya se redujo el tamaño de la matriz//aqui dentro esta el n--
        //cout << "creada nueva matriz de distancias" << endl;
    }
    //al final se calculo una nuev matriz de distancias, por lo tanto arreglo ID tiene solo 2 elementos
    //UNION: a ellos los unimos
    //cout << "final de las iteracion hasta n-2" << endl;
    //cout << "distancias" << endl;
    //ImprimirArreglo(ArregloId, DimensionMatrizI);
    //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
    //cout << "distancias modificadas" << endl;
    //ImprimirArreglo(ArregloId, DimensionMatrizI);
    //ImprimirMatriz(MatrizDistanciasModificadas, DimensionMatrizI);
    Nodos[ArregloId[0]]->Padre = Nodos[ArregloId[1]];
    Nodos[ArregloId[0]]->PadreId = ArregloId[1];//o usando -D sobre la linea anterior
    Nodos[ArregloId[0]]->DistanciaPadre= MatrizDistancias[0][1];//o usando -D sobre la linea anterior
    Nodos[ArregloId[1]]->Padre = Nodos[ArregloId[0]];
    Nodos[ArregloId[1]]->PadreId = ArregloId[0];//o usando -D sobre la linea anterior
    Nodos[ArregloId[1]]->DistanciaPadre= MatrizDistancias[0][1];//o usando -D sobre la linea anterior
    //falta calcular estas distancias
    Arbol = Nodos;
    for(int i = 0; i < NumeroNodosReales; i++){
        hipFree(MatrizDistanciasHost[i]);
    }
    hipFree(MatrizDistanciasDevice);
    hipFree(DivergenciasDevice);
    return NumeroNodos;
}

float NJ::Mij(int i, int j){// en espacio de arreglo id 
    //como trabajo sobre la matriz inferior, esta bien lo i j que recibo, que me marque el (1,0)
    return MatrizDistancias[i][j] - (Divergencias[i] + Divergencias[j])/(DimensionMatrizI-2);
}

void NJ::NuevaMatrizDistancias(int i, int j, float ** MatrizDistanciasDevice, float ** MatrizDistanciasHost){//i y j son los nodos que fueron desginados como similares en esa iteracion
    //antes de ahcer el corrimiento seria buno calcular las deistancias para el nuevo, y solo hacer el cambio con i
    //como el j es el mas a la izquierda, ese lo dejo, y muevo el i hacia el final de la matriz
    //Host solo almacena los punteros de las filas en device
    //cout << "Nueva matriz de distacnias " << i << ", " << j << endl;//se escojo e primero
    float Distanciaij = MatrizDistancias[i][j];//este se puede dejar asi, ya que uso la matriz inferior

    for(int k = i+1; k < DimensionMatrizI; k++){
        int tempId = ArregloId[k-1];//solo debe modificar lo concerniete al arreglo j
        ArregloId[k-1] = ArregloId[k];//a la vez que cambio debo actulizar sus poseisiones en el arregloID
        ArregloId[k] = tempId;
        
    }
    //estos i y j, estan sobre el ArrgloId
    MoviendoIDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, DimensionMatrizI, i);
    MoviendoJDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, DimensionMatrizI, i);
    //cout << "i al final" << endl;
    //ImprimirArreglo(ArregloId, DimensionMatrizI);
    //ImprimirMatriz(MatrizDistancias, DimensionMatrizI);
    DimensionMatrizI--;
    //j ahora esta en la posicion DimensionMatriz
    ActualizarDistanciasDevice<<<ceil(DimensionMatrizI/256.0), 256>>>(MatrizDistanciasDevice, DimensionMatrizI, j, Distanciaij);
    int size = DimensionMatrizI * sizeof(float);
    for(int i = 0; i < DimensionMatrizI; i++){
        hipMemcpy(MatrizDistancias[i], MatrizDistanciasHost[i], size, hipMemcpyDeviceToHost);
    }
    //cout << "fin creacio nnueva matriz" << endl;
    //en que momento ubico al nuevo nodo viertual cundo lo creo?
    //
}

void NJ::ImprimirMatriz(float ** Matriz, int n){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            printf("%.3f\t", Matriz[i][j]);
        }
        printf("\n");
    }
}

void NJ::ImprimirArreglo(int * Arreglo, int n){
    for(int i = 0; i < n; i++){
        printf("%d\t", ArregloId[i]);
    }
    printf("\n");
}

void NJ::ImprimirDivergencias(float * Divergencia, int n){
    printf("Divergencias:\n");
    for(int i = 0; i < n; i++){
        printf("%.3f\t", Divergencia[i]);
    }
    printf("\n");
}

void NJ::CrearNodoVirtual(int i, int j){//es que deberia recibir algun dato, nombre del documento quiza, para los reales
    //que pasa si quiero crear mas nodos reales? mm en teoria no puedo, ya que habira que recalcular todo
    //creo que esto sera para crear nodos virtuales
    ////recibe los i y j del arreglo de nodos
    //aqui recibo los i y j siendo i > j, por lo tanto lo que hacia con j debo hacerlo con y y lo que hacia con i debo hacerlo conj
    //float Lj = MatrizDistancias[j][i]/2 + (Divergencias[j] - Divergencias[i])/(2*(DimensionMatrizI-2));
    //float Li = MatrizDistancias[j][i] - Lj;
    //cout << "entrada: " << i << "-" << j << endl;
    //cout << MatrizDistancias[i][j] << endl;
    //cout << Divergencias[j] << endl;
    //cout << Divergencias[i] << endl;
    float Lj = MatrizDistancias[i][j]/2 + (Divergencias[j] - Divergencias[i])/(2*(DimensionMatrizI-2));
    float Li = MatrizDistancias[i][j] - Lj;
    i = ArregloId[i];
    j = ArregloId[j];

    int nv = NumeroNodosReales + NumeroNodosVirtuales;
    //cout <<"arreglo: " << i << "-" << j << " " << "id nuevo nodo virtual " << nv << endl;
    //cout << "Lj: " << Lj << " Li: " << Li << endl;

    //Nodos[nv]->Id = nv;
    Nodos[nv]->HijosId[0] = j;
    Nodos[nv]->HijosId[1] = i;
    Nodos[nv]->Hijos[0] = Nodos[j];
    Nodos[nv]->Hijos[1] = Nodos[i];
    Nodos[nv]->DistanciasHijos[0] = Lj;
    Nodos[nv]->DistanciasHijos[1] = Li;
    Nodos[nv]->Orden = 0;//iteracion
    Nodos[j]->Padre = Nodos[nv];
    Nodos[j]->PadreId = Nodos[nv]->Id;//o odria ser solo NumeroNodos
    Nodos[j]->DistanciaPadre = Lj;
    Nodos[j]->Orden = Orden;//iteracion
    Orden++;
    Nodos[i]->Padre = Nodos[nv];
    Nodos[i]->PadreId = Nodos[nv]->Id;//o odria ser solo NumeroNodos
    Nodos[i]->DistanciaPadre = Li;
    Nodos[i]->Orden = Orden;//iteracion
    Orden++;
    NumeroNodosVirtuales++;

    //falta agregar lo de las ditancias de los branchs
//afuera de esta funcon se maneja lo de las hojas para los nodos virtaules, 
}

//mejorar las contruscciones de los nodos por defecto y demas, para hacer mas eficiente esta arte
//y si uso la clase vector?? contruir una version que lo use
void NJ::DatosIniciales(string * Datos, int n){
    //en teoria siempre se necesitan la misma cantidad de nodos virtuales, ya que el algoritmos siempre da las iteraciones fijas, solo depende de si n es par o impar;
    //por lo lanto se pueden crear de una vez los nodos virtuales, y solo llenarlos con la impofrmacion pertinenete y determnando a lcula llear, en cada iteracion, en lugar de estar creandolos en cada iteracion
    //crear los nodos reales
    NumeroNodos = n + n - 2;
    NumeroNodosReales = n;
    Nodos = new Nodo * [NumeroNodos];
    for(int i = 0; i < NumeroNodosReales; i++){
        Nodos[i] = new Nodo;
        Nodos[i]->Id = i;
        Nodos[i]->Valido = 1;//si sera virtual o real
        Nodos[i]->Nombre = Datos[i];
    }
    for(int i = NumeroNodosReales; i < NumeroNodos; i++){//para crear los nodos virtuales
        Nodos[i] = new Nodo;
        Nodos[i]->Id = i;
        Nodos[i]->Valido = 0;//si sera virtual o real
    }
    NumeroNodosVirtuales = 0;//aun no los he creado coomo parte del algoritmo,
    //esta variable me ayudara a iterar para trabajr sobre le nodo virtual aporpiado,
    //lo que nates hacian cuando incrementaba la variable NUmeroNodos en cada iteracion
}

NJ::NJ(){

}

NJ::~NJ(){
    for(int i = 0; i < NumeroNodosReales; i++){//quiza sea util si la matriz no esta de la forma que deseo
        delete [] MatrizDistancias[i];
    }
    delete [] MatrizDistancias;
    delete [] ArregloId;
    delete [] Divergencias;
    //el arreglo nodos debo eliminarlo??
}

//buscar donde hay redundancia en el acceso a datos, quiza todos los sij, se puedan calcular a la vez, o cosas asi
//busar como manejar en la misma matriz los nuevos nodos virtuales, los reemplazos etc, para no crear matrices mas grandes, y seguir utilizando la que esta, quiza ids temporanoles, o que se yo
////la cantidad de iteraciones es fija, por lo tanto crear los nodos virtuales necesarios desde el inicio, y en la funcion CrearNodoVIrual solamente hacer las uniones y calculos respectivos
